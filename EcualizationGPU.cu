#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "common.h"
#include <hip/hip_runtime.h>

#include <chrono>

using namespace std;
using namespace cv;

__global__ void histog_kernel(unsigned char* input, unsigned char* output, int width, int height, int grayWidthStep, long totalSize){

    //2D Index of current thread
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    unsigned int nxy = threadIdx.x + threadIdx.y * blockDim.x;

    //Location of gray pixel in output
    const int gray_tid  = iy * grayWidthStep + ix;

    __shared__ int hist[256];
    __shared__ int hist_s[256];

    hist[nxy] = 0;

    __syncthreads();
    
    //Only valid threads perform memory I/O
	if((ix<width) && (iy<height))
	{
        atomicAdd(&hist[input[gray_tid]], 1);
    }

    __syncthreads();


    if(nxy < 256 && blockIdx.x == 0 && blockIdx.y == 0)
	{
		__syncthreads();

		for(int i = 0; i <= nxy; i++){

           hist_s[nxy] += hist[i];

           printf("%d\n", hist[i]);

        }
    }
    __syncthreads();

    if(nxy < 256 && blockIdx.x == 0 && blockIdx.y==0){

        int aux = (hist_s[nxy]*255)/totalSize;

		hist_s[nxy] = aux; 
	}

    __syncthreads();

    if((ix < width) && (iy < height))
	{
		int Index = input[gray_tid];
		output[gray_tid] = hist_s[Index];
    }

}

void histog(const cv::Mat& input, cv::Mat& output)
{
	//Calcu late total number of bytes, input and output image are both gray scale
	const int grayBytes = output.step * output.rows;

	unsigned char *d_input, *d_output;

	//Allocate device memory
	SAFE_CALL(hipMalloc((void**)&d_input,grayBytes),"CUDA Malloc Failed");
	SAFE_CALL(hipMalloc((void**)&d_output,grayBytes),"CUDA Malloc Failed");

	//Copy data from OpenCV input image to device memory
	SAFE_CALL(hipMemcpy(d_input, input.ptr(), grayBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");
    SAFE_CALL(hipMemcpy(d_output, output.ptr(), grayBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");

	//Specify a reasonable block size
	const dim3 block(16,16);

	//Calculate grid size to cover the whole image
	const dim3 grid((input.cols)/block.x, (input.rows)/block.y);

	//Launch the color conversion kernel
	histog_kernel<<<grid,block>>>(d_input,d_output,input.cols,input.rows,input.step, grayBytes);

	//Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");

	//Copy back data from destination device meory to OpenCV output image
	SAFE_CALL(hipMemcpy(output.ptr(),d_output,grayBytes,hipMemcpyDeviceToHost),"CUDA Memcpy Host To Device Failed");

	//Free the device memory
	SAFE_CALL(hipFree(d_input),"CUDA Free Failed");
	SAFE_CALL(hipFree(d_output),"CUDA Free Failed");
}

int main (int argc, char** argv){


    if (argc < 2){
         
        cout << "No hay argumentos suficientes" << endl;
    }
    else {
        
        Mat image;
        Mat grayImage(image.rows,image.cols,CV_8UC1);
        Mat output(image.rows,image.cols,CV_8UC1);

        image = imread(argv[1], CV_LOAD_IMAGE_COLOR);

        cout << "Image size Step: "<< image.step << " Rows: " << image.rows << " Cols: " << image.cols << endl;

        cvtColor(image, grayImage, CV_BGR2GRAY);

        output = grayImage.clone();

        
        histog(grayImage, output);      

        //Allow the windows to resize
        namedWindow("Input", cv::WINDOW_NORMAL);
        namedWindow("Output", cv::WINDOW_NORMAL);

        //Show the input and output
        imshow("Input", grayImage);
        imshow("Output", output);  
            
    }   

    waitKey(0);
    return 0;

}